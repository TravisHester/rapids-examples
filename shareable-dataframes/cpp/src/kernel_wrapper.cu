/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudf/utilities/type_dispatcher.hpp"
#include "kernel_wrapper.hpp"
 
 CudfWrapper::CudfWrapper(cudf::mutable_table_view table_view) {
   mtv = table_view;
 }
 
 void CudfWrapper::tenth_mm_to_inches(int column_index) {
 
  // Example of showing num_columns and num_rows only for potential debugging
  printf("kernel_wrapper.cu # of columns: %lu\n", mtv.num_columns());
  printf("kernel_wrapper.cu # of rows: %lu\n", mtv.num_rows());
 
  std::unique_ptr<cudf::mutable_column_device_view, std::function<void(cudf::mutable_column_device_view*)>> 
        mutable_device_column = cudf::mutable_column_device_view::create(mtv.column(column_index));

  printf("Number rows in mutable_device_column: %lu\n", mutable_device_column->size());

  auto s = cudf::get_element(mtv.column(column_index), 0);
  using ScalarType = cudf::scalar_type_t<int64_t>;
  auto typed_s     = static_cast<ScalarType const *>(s.get());
  printf("Value before kernel: %d\n", typed_s->value());
 
  // Invoke the Kernel to convert tenth_mm -> inches
  kernel_tenth_mm_to_inches<<<1, 1>>>(*mutable_device_column);
  hipError_t err = hipStreamSynchronize(0);
  printf("hipStreamSynchronize Response = %d\n", (int)err);

  s = cudf::get_element(mtv.column(column_index), 0);
  typed_s     = static_cast<ScalarType const *>(s.get());
  printf("Value after kernel: %d\n", typed_s->value()); // Value is same as before
 }
 
 CudfWrapper::~CudfWrapper() {
   // It is important to note that CudfWrapper does not own the underlying Dataframe 
   // object and that will be freed by the Python/Cython layer later.
 }
